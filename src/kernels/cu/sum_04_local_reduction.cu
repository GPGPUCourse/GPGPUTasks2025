#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/shared_device_buffer.h>
#include <libgpu/work_size.h>

#include <libgpu/cuda/cu/common.cu>

#include "../defines.h"

#define WARP_SIZE 32

__global__ void sum_04_local_reduction(
    const unsigned int* a,
    unsigned int* b,
    unsigned int n)
{
    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    const uint local_index = threadIdx.x;
    __shared__ unsigned int local_data[GROUP_SIZE];
    if (index < n) {
        local_data[local_index] = 0;
        local_data[local_index] += a[index];
    } else {
        local_data[local_index] = 0;
    }
    __syncthreads();
    if (local_index == 0) {
        unsigned int local_sum = 0;
        for (int i = 0; i < GROUP_SIZE; ++i) {
            local_sum += local_data[i];
        }
        b[index / GROUP_SIZE] = local_sum;
    }
}

namespace cuda {
void sum_04_local_reduction(const gpu::WorkSize& workSize,
    const gpu::gpu_mem_32u& a, gpu::gpu_mem_32u& sum, unsigned int n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 6573652345243, context.type());
    hipStream_t stream = context.cudaStream();
    ::sum_04_local_reduction<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(a.cuptr(), sum.cuptr(), n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
