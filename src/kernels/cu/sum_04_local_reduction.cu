#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "../defines.h"

#define WARP_SIZE 32

__global__ void sum_04_local_reduction(
    const unsigned int* a,
    unsigned int* b,
    unsigned int  n)
{
    // Подсказки:
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int local_index = threadIdx.x;
    __shared__ unsigned local_data[GROUP_SIZE];
    local_data[local_index] = (index >= n) ? 0 : a[index];
    __syncthreads();
    if (local_index==0) {
        unsigned int local_sum = 0;
        for (unsigned int i = 0; i < GROUP_SIZE; ++i) {
            local_sum += local_data[i];
        }
        b[blockIdx.x] = local_sum;
    }
    // TODO
}

namespace cuda {
void sum_04_local_reduction(const gpu::WorkSize &workSize,
    const gpu::gpu_mem_32u &a, gpu::gpu_mem_32u &sum, unsigned int n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 6573652345243, context.type());
    hipStream_t stream = context.cudaStream();
    ::sum_04_local_reduction<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(a.cuptr(), sum.cuptr(), n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
