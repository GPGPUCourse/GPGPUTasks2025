#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "../defines.h"

#define WARP_SIZE 32

__global__ void sum_04_local_reduction(
    const unsigned int* a,
    unsigned int* b,
    unsigned int  n)
{
    // Подсказки:
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int local_index = threadIdx.x;
    __shared__ unsigned int local_data[GROUP_SIZE];

    if (index < n) {
        local_data[local_index] = a[index];
    } else {
        local_data[local_index] = 0;
    }

    __syncthreads();

    if (local_index == 0) {
        unsigned int local_sum = 0;
        for (const unsigned int i : local_data) {
            local_sum += i;
        }
        b[index / GROUP_SIZE] = local_sum;
    }
}

namespace cuda {
void sum_04_local_reduction(const gpu::WorkSize &workSize,
    const gpu::gpu_mem_32u &a, gpu::gpu_mem_32u &sum, unsigned int n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 6573652345243, context.type());
    hipStream_t stream = context.cudaStream();
    ::sum_04_local_reduction<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(a.cuptr(), sum.cuptr(), n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
