#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "helpers/rassert.cu"
#include "../defines.h"

__global__ void matrix_transpose_naive(
                       const float* matrix,            // w x h
                             float* transposed_matrix, // h x w
                             unsigned int w,
                             unsigned int h)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < w && y < h) {
        transposed_matrix[x * h + y] = matrix[y * w + x];
    }
}

namespace cuda {
void matrix_transpose_naive(const gpu::WorkSize &workSize,
            const gpu::gpu_mem_32f &matrix, gpu::gpu_mem_32f &transposed_matrix, unsigned int w, unsigned int h)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::matrix_transpose_naive<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(matrix.cuptr(), transposed_matrix.cuptr(), w, h);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
