#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/shared_device_buffer.h>
#include <libgpu/work_size.h>

#include <libgpu/cuda/cu/common.cu>

#include "../defines.h"
#include "helpers/rassert.cu"

__device__ __forceinline__ void set(float* a, int n, int m, int i, int j, float x)
{
    if (i < n && j < m) {
        a[i * m + j] = x;
    }
}

__device__ __forceinline__ float get(const float* a, int n, int m, int i, int j)
{
    return (i < n && j < m) ? a[i * m + j] : 0.0f;
}

__global__ void matrix_transpose_naive(
    const float* matrix, // h x w
    float* transposed_matrix, // w x h
    unsigned int w,
    unsigned int h)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x; // 0..w
    int i = blockIdx.y * blockDim.y + threadIdx.y; // 0..h

    float data = get(matrix, h, w, i, j);
    set(transposed_matrix, w, h, j, i, data);
}

namespace cuda {
void matrix_transpose_naive(const gpu::WorkSize& workSize,
    const gpu::gpu_mem_32f& matrix, gpu::gpu_mem_32f& transposed_matrix, unsigned int w, unsigned int h)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::matrix_transpose_naive<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(matrix.cuptr(), transposed_matrix.cuptr(), w, h);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
