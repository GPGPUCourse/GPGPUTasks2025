#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "helpers/rassert.cu"
#include "../defines.h"

// Include WMMA header with nvcuda::wmma namespace
#include <mma.h>
using namespace nvcuda;

__global__ void matrix_multiply_wmma(
                       const float* a, // rows=h x cols=k
                       const float* b, // rows=k x cols=w
                             float* c, // rows=h x cols=w
                       unsigned int w,
                       unsigned int h,
                       unsigned int k)
{
    constexpr unsigned int WMMA_M = 16, WMMA_N = 16, WMMA_K = 16;
    constexpr unsigned int BLOCK_M = 32, BLOCK_N = 32;
    constexpr unsigned int as_size = BLOCK_M * WMMA_K, bs_size = WMMA_K * BLOCK_N;
    const unsigned int threads_per_block = blockDim.x * blockDim.y;
    const unsigned int thrd = threadIdx.y * blockDim.x + threadIdx.x;
    const unsigned int warp = thrd / warpSize;
    const unsigned int warp_row = warp / 2;
    const unsigned int warp_col = warp % 2;
    const unsigned int row_start = blockIdx.y * BLOCK_M;
    const unsigned int col_start = blockIdx.x * BLOCK_N;
    const unsigned int res_row_start = row_start + warp_row * WMMA_M;
    const unsigned int res_col_start = col_start + warp_col * WMMA_N;
    
    __shared__ half as[as_size];
    __shared__ half bs[bs_size];
    const half* as_ptr = &as[warp_row * WMMA_M * WMMA_K];
    const half* bs_ptr = &bs[warp_col * WMMA_N * WMMA_K];
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
    wmma::fill_fragment(c_frag, 0.f);

    #pragma unroll
    for (unsigned int i = 0; i < k; i += WMMA_K) {
        #pragma unroll
        for (unsigned int t = thrd; t < as_size; t += threads_per_block) {
            unsigned int row = t / WMMA_K;
            unsigned int col = t % WMMA_K;
            unsigned int a_row = row_start + row;
            unsigned int a_col = i + col;
            float val = 0;
            if (a_row < h && a_col < k)
                val = __ldg(&a[a_row * k + a_col]);
            as[row * WMMA_K + col] = __float2half_rn(val);
        }

        #pragma unroll
        for (unsigned int t = thrd; t < bs_size; t += threads_per_block) {
            unsigned int row = t / BLOCK_N;
            unsigned int col = t % BLOCK_N;
            unsigned int b_row = i + row;
            unsigned int b_col = col_start + col;
            float val = 0;
            if (b_row < k && b_col < w)
                val = __ldg(&b[b_row * w + b_col]);
            bs[row + col * WMMA_K] = __float2half_rn(val);
        }

        __syncthreads();

        wmma::load_matrix_sync(a_frag, as_ptr, WMMA_K);
        wmma::load_matrix_sync(b_frag, bs_ptr, WMMA_K);
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

        __syncthreads();
    }

    wmma::store_matrix_sync(c + res_row_start * w + res_col_start, c_frag, w, wmma::mem_row_major);
}

namespace cuda {
void matrix_multiply_wmma(const gpu::WorkSize &workSize,
            const gpu::gpu_mem_32f &a, const gpu::gpu_mem_32f &b, gpu::gpu_mem_32f &c, unsigned int w, unsigned int h, unsigned int k)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::matrix_multiply_wmma<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(a.cuptr(), b.cuptr(), c.cuptr(), w, h, k);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda

