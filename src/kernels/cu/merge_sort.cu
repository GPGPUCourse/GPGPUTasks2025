#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "helpers/rassert.cu"
#include "../defines.h"

__global__ void merge_sort(
    const unsigned int* input_data,
          unsigned int* output_data,
                   int  sorted_k,
                   int  n)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    // TODO
}

namespace cuda {
void merge_sort(const gpu::WorkSize &workSize,
            const gpu::gpu_mem_32u &input_data, gpu::gpu_mem_32u &output_data, int sorted_k, int n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::merge_sort<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(input_data.cuptr(), output_data.cuptr(), sorted_k, n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
