#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "helpers/rassert.cu"
#include "../defines.h"

#define uint unsigned int

__global__ void radix_sort_03_scan_accumulation(
    const uint* buffer_fenwick_gpu,
          uint* prefix_sum_accum_gpu,
    unsigned int n)
{
    const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    uint global_sum = 0;
    uint global_idx = idx / GROUP_SIZE;
    if (global_idx > 0) {
        for (; global_idx > 0; global_idx -= global_idx & -global_idx) {
            global_sum += buffer_fenwick_gpu[global_idx - 1];
        }
    }

    prefix_sum_accum_gpu[idx] += global_sum;
}

namespace cuda {
void radix_sort_03_scan_accumulation(const gpu::WorkSize &workSize,
            const gpu::gpu_mem_32u &buffer_fenwick_gpu, gpu::gpu_mem_32u &prefix_sum_accum_gpu, uint n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::radix_sort_03_scan_accumulation<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(buffer_fenwick_gpu.cuptr(), prefix_sum_accum_gpu.cuptr(), n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda