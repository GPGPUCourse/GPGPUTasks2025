#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "../defines.h"

__global__ void sum_03_local_memory_atomic_per_workgroup(
    const unsigned int* a,
    unsigned int* sum,
    unsigned int  n)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int local_index = threadIdx.x;
    __shared__ unsigned int local_data[GROUP_SIZE];

    if (index < n) {
        local_data[local_index] = a[index];
    } else {
        local_data[local_index] = 0;
    }

    __syncthreads();
    if (index < n && local_index == 0) {
        unsigned int temp_sum = 0;
        for (unsigned int i = 0; i < GROUP_SIZE; ++i) {
            temp_sum += local_data[i];
        }
        atomicAdd(sum, temp_sum);
    }
}

namespace cuda {
void sum_03_local_memory_atomic_per_workgroup(const gpu::WorkSize &workSize,
    const gpu::gpu_mem_32u &a, gpu::gpu_mem_32u &sum, unsigned int n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 6573652345243, context.type());
    hipStream_t stream = context.cudaStream();
    ::sum_03_local_memory_atomic_per_workgroup<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(a.cuptr(), sum.cuptr(), n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
