#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "../defines.h"

__global__ void sum_03_local_memory_atomic_per_workgroup(
    const unsigned int* a,
    unsigned int* sum,
    unsigned int  n)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int local_index = threadIdx.x;
    __shared__ unsigned int local_data[GROUP_SIZE];

    if (index < n / LOAD_K_VALUES_PER_ITEM) {
        unsigned int my_sum = 0;
        for (unsigned int i = 0; i < LOAD_K_VALUES_PER_ITEM; ++i) {
            my_sum += a[i * (n/LOAD_K_VALUES_PER_ITEM) + index];
        }
        local_data[local_index] = my_sum;
    } else {
        local_data[local_index] = 0;
    }

    __syncthreads();

    if (local_index == 0) {
        unsigned int my_sum = 0;
        for (unsigned int i = 0; i < GROUP_SIZE; ++i) {
            my_sum += local_data[i];
        }
        atomicAdd(sum, my_sum);
    }
}

namespace cuda {
void sum_03_local_memory_atomic_per_workgroup(const gpu::WorkSize &workSize,
    const gpu::gpu_mem_32u &a, gpu::gpu_mem_32u &sum, unsigned int n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 6573652345243, context.type());
    hipStream_t stream = context.cudaStream();
    ::sum_03_local_memory_atomic_per_workgroup<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(a.cuptr(), sum.cuptr(), n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
