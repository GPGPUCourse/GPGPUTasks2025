#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "../defines.h"

__global__ void sum_03_local_memory_atomic_per_workgroup(
    const unsigned int* a,
    unsigned int* sum,
    unsigned int  n)
{
    // Подсказки:
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int local_index = threadIdx.x;
    __shared__ unsigned int local_data[GROUP_SIZE];
    // __syncthreads();

    // TODO
    if (index < n)
        local_data[local_index] = a[index];
    else 
        local_data[local_index] = 0;
    __syncthreads();

    if (local_index == 0) {
        unsigned int my_sum = 0;
        for (unsigned int i = 0; i < GROUP_SIZE; ++i) {
            my_sum += local_data[i];
        }
        atomicAdd(sum, my_sum);
    }
}

namespace cuda {
void sum_03_local_memory_atomic_per_workgroup(const gpu::WorkSize &workSize,
    const gpu::gpu_mem_32u &a, gpu::gpu_mem_32u &sum, unsigned int n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 6573652345243, context.type());
    hipStream_t stream = context.cudaStream();
    ::sum_03_local_memory_atomic_per_workgroup<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(a.cuptr(), sum.cuptr(), n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
