#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/shared_device_buffer.h>
#include <libgpu/work_size.h>

#include <libgpu/cuda/cu/common.cu>

#include "../defines.h"

__global__ void sum_03_local_memory_atomic_per_workgroup(
    const unsigned int* a,
    unsigned int* sum,
    unsigned int n)
{
    // Подсказки:
    // const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    // const uint local_index = threadIdx.x;
    // __shared__ unsigned int local_data[GROUP_SIZE];
    // __syncthreads();

    const uint index = blockIdx.x * blockDim.x + threadIdx.x;
    const uint local_index = threadIdx.x;
    __shared__ unsigned int local_data[GROUP_SIZE];

    unsigned int partial_sum = 0;

    for (uint i = index; i < n; i += blockDim.x * gridDim.x) {
        partial_sum += a[i];
    }

    local_data[local_index] = partial_sum;
    __syncthreads();

    for (uint s = blockDim.x / 2; s > 0; s /= 2) {
        if (local_index < s) {
            local_data[local_index] += local_data[local_index + s];
        }
        __syncthreads();
    }

    if (local_index == 0) {
        atomicAdd(sum, local_data[0]);
    }
}

namespace cuda {
void sum_03_local_memory_atomic_per_workgroup(const gpu::WorkSize& workSize,
    const gpu::gpu_mem_32u& a, gpu::gpu_mem_32u& sum, unsigned int n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 6573652345243, context.type());
    hipStream_t stream = context.cudaStream();
    ::sum_03_local_memory_atomic_per_workgroup<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(a.cuptr(), sum.cuptr(), n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
