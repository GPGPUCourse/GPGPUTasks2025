#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "../defines.h"

__global__ void sum_03_local_memory_atomic_per_workgroup(
    const unsigned int* a,
    unsigned int* sum,
    unsigned int  n)
{
    __shared__ unsigned int local_data[GROUP_SIZE];
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int local_index = threadIdx.x;

    unsigned int my_sum = 0;
    if (index < n / LOAD_K_VALUES_PER_ITEM) {
        for (unsigned int i = 0; i < LOAD_K_VALUES_PER_ITEM; ++i) {
            my_sum += a[i * (n/LOAD_K_VALUES_PER_ITEM) + index];
        }
    }
    local_data[local_index] = my_sum;
    __syncthreads();

    if (local_index == 0) {
        for (unsigned int i = 1; i < GROUP_SIZE; ++i) {
            my_sum += local_data[i];
        }
        atomicAdd(sum, my_sum);
    }
}

namespace cuda {
void sum_03_local_memory_atomic_per_workgroup(const gpu::WorkSize &workSize,
    const gpu::gpu_mem_32u &a, gpu::gpu_mem_32u &sum, unsigned int n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 6573652345243, context.type());
    hipStream_t stream = context.cudaStream();
    ::sum_03_local_memory_atomic_per_workgroup<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(a.cuptr(), sum.cuptr(), n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
