#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "helpers/rassert.cu"
#include "../defines.h"

__global__ void prefix_sum_01_sum_reduction(
    // это лишь шаблон! смело меняйте аргументы и используемые буфера! можете сделать даже больше кернелов, если это вызовет затруднения - смело спрашивайте в чате
    // НЕ ПОДСТРАИВАЙТЕСЬ ПОД СИСТЕМУ! СВЕРНИТЕ С РЕЛЬС!! БУНТ!!! АНТИХАЙП!11!!1
    const unsigned int* pow2_sum, // contains n values
          unsigned int* next_pow2_sum, // will contain (n+1)/2 values
    unsigned int n)
{
    // TODO
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ unsigned int buff[GROUP_SIZE];

    buff[threadIdx.x] = pow2_sum[index << 1];
    if ((index << 1) + 1 < n) {
        buff[threadIdx.x] += pow2_sum[(index << 1) + 1];
    }
    __syncthreads();

    if (index < n)
        next_pow2_sum[index] = buff[threadIdx.x];
}

namespace cuda {
void prefix_sum_01_sum_reduction(const gpu::WorkSize &workSize,
            const gpu::gpu_mem_32u &pow2_sum, gpu::gpu_mem_32u &next_pow2_sum, unsigned int n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::prefix_sum_01_sum_reduction<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(pow2_sum.cuptr(), next_pow2_sum.cuptr(), n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
