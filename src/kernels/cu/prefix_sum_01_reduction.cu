#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "helpers/rassert.cu"
#include "../defines.h"

__global__ void prefix_sum_01_sum_reduction(
    // это лишь шаблон! смело меняйте аргументы и используемые буфера! можете сделать даже больше кернелов, если это вызовет затруднения - смело спрашивайте в чате
    // НЕ ПОДСТРАИВАЙТЕСЬ ПОД СИСТЕМУ! СВЕРНИТЕ С РЕЛЬС!! БУНТ!!! АНТИХАЙП!11!!1
    const unsigned int* pow2_sum, // contains n values
          unsigned int* next_pow2_sum, // will contain (n+1)/2 values
    unsigned int n)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int previous_index = (index << 1);
    if (previous_index >= n)  return;
    next_pow2_sum[index] = pow2_sum[previous_index];
    if (previous_index + 1 >= n) return;
    next_pow2_sum[index] += pow2_sum[previous_index + 1];
}

namespace cuda {
void prefix_sum_01_sum_reduction(const gpu::WorkSize &workSize,
            const gpu::gpu_mem_32u &pow2_sum, gpu::gpu_mem_32u &next_pow2_sum, unsigned int n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::prefix_sum_01_sum_reduction<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(pow2_sum.cuptr(), next_pow2_sum.cuptr(), n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
