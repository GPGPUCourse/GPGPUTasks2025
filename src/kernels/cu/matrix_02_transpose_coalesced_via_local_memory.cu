#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "helpers/rassert.cu"
#include "../defines.h"

__global__ void matrix_transpose_coalesced_via_local_memory(
                       const float* matrix,            // w x h
                             float* transposed_matrix, // h x w
                             unsigned int w,
                             unsigned int h)
{
    __shared__ float tile[GROUP_SIZE_X][GROUP_SIZE_X + 1];
    unsigned int i = blockIdx.x * GROUP_SIZE_X + threadIdx.x;
    unsigned int j = blockIdx.y * GROUP_SIZE_X + threadIdx.y;
    if (i >= w)
        return;

    #pragma unroll
    for (unsigned int offset = 0; offset < GROUP_SIZE_X; offset += GROUP_SIZE_Y) {
        const unsigned int y = j + offset;
        if (y >= h)
            break;

        tile[threadIdx.y + offset][threadIdx.x] = matrix[y * w + i];
    }

    __syncthreads();

    i = blockIdx.y * GROUP_SIZE_X + threadIdx.x;
    j = blockIdx.x * GROUP_SIZE_X + threadIdx.y;
    if (i >= h)
        return;

    #pragma unroll
    for (unsigned int offset = 0; offset < GROUP_SIZE_X; offset += GROUP_SIZE_Y) {
        const unsigned int y = j + offset;
        if (y >= w)
            return;

        transposed_matrix[y * h + i] = tile[threadIdx.x][threadIdx.y + offset];
    }
}

namespace cuda {
void matrix_transpose_coalesced_via_local_memory(const gpu::WorkSize &workSize,
            const gpu::gpu_mem_32f &matrix, gpu::gpu_mem_32f &transposed_matrix, unsigned int w, unsigned int h)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::matrix_transpose_coalesced_via_local_memory<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(matrix.cuptr(), transposed_matrix.cuptr(), w, h);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
