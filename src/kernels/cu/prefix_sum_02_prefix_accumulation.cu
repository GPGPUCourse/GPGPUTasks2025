#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "helpers/rassert.cu"
#include "../defines.h"

__global__ void prefix_sum_02_prefix_accumulation(
    // это лишь шаблон! смело меняйте аргументы и используемые буфера! можете сделать даже больше кернелов, если это вызовет затруднения - смело спрашивайте в чате
    // НЕ ПОДСТРАИВАЙТЕСЬ ПОД СИСТЕМУ! СВЕРНИТЕ С РЕЛЬС!! БУНТ!!! АНТИХАЙП!11!!1
    const unsigned int* pow2_sum, // pow2_sum[i] = sum[i*2^pow2; (i+1)*2^pow2)
          unsigned int* prefix_sum_accum, // we want to make it finally so that prefix_sum_accum[i] = sum[0, i]
    unsigned int n,
    unsigned int pow2) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx & (1 << pow2)) {
        const unsigned int pow2_idx = idx / (1 << pow2) - 1;
        prefix_sum_accum[idx] += pow2_sum[pow2_idx];
    }
}

namespace cuda {
void prefix_sum_02_prefix_accumulation(const gpu::WorkSize &workSize,
            const gpu::gpu_mem_32u &pow2_sum, gpu::gpu_mem_32u &prefix_sum_accum, unsigned int n, unsigned int pow2)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::prefix_sum_02_prefix_accumulation<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(pow2_sum.cuptr(), prefix_sum_accum.cuptr(), n, pow2);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
