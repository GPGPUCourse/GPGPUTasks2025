#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/shared_device_buffer.h>
#include <libgpu/work_size.h>

#include <libgpu/cuda/cu/common.cu>

#include "../defines.h"
#include "helpers/rassert.cu"

__global__ void copy_buffer(
    // это лишь шаблон! смело меняйте аргументы и используемые буфера! можете сделать даже больше кернелов, если это вызовет затруднения - смело спрашивайте в чате
    // НЕ ПОДСТРАИВАЙТЕСЬ ПОД СИСТЕМУ! СВЕРНИТЕ С РЕЛЬС!! БУНТ!!! АНТИХАЙП!11!!1
    const unsigned int* from,
    unsigned int* to,
    unsigned int n)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    to[index] = from[index];
}

namespace cuda {
void copy_buffer(const gpu::WorkSize& workSize, const gpu::gpu_mem_32u& from, gpu::gpu_mem_32u& to, unsigned int n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::copy_buffer<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(from.cuptr(), to.cuptr(), n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
