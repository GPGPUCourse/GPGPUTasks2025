#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "helpers/rassert.cu"
#include "../defines.h"

#define uint unsigned int

__global__ void radix_sort_01_scan_build_fenwick(
    const uint* input,
          uint* output,
    uint n,
    uint d,
    uint bit)
{
    __shared__ uint part[GROUP_SIZE];

    const uint bl_offset = (blockIdx.x * blockDim.x) << (d * 8);
    const uint g_idx = ((blockIdx.x * blockDim.x + threadIdx.x + 1) << (d * 8)) - 1;
    const uint idx = threadIdx.x;
    const uint l_size = min(GROUP_SIZE, n - bl_offset);

    if (g_idx < n) {
        if (d == 0) {
            part[idx] = (input[g_idx] >> bit) & 1;
        } else {
            part[idx] = input[g_idx];
        }

        __syncthreads();

#pragma unroll
        for (uint i = 1; i <= G_DEPTH; ++i) {
            const uint step = (1 << i);

            const uint j1 = (idx + 1) * step - 1;
            const uint j0 = j1 - (step >> 1);
            if (j1 < l_size) {
                part[j1] += part[j0];
            }

            __syncthreads();
        }

        output[g_idx] = part[idx];
    }
}

namespace cuda {
void radix_sort_01_scan_build_fenwick(const gpu::WorkSize &workSize,
            const gpu::gpu_mem_32u &input, gpu::gpu_mem_32u &output, uint n, uint d, uint bit)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::radix_sort_01_scan_build_fenwick<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(input.cuptr(), output.cuptr(), n, d, bit);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda