#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "helpers/rassert.cu"
#include "../defines.h"

__global__ void radix_sort_01_local_counting(
    // это лишь шаблон! смело меняйте аргументы и используемые буфера! можете сделать даже больше кернелов, если это вызовет затруднения - смело спрашивайте в чате
    // НЕ ПОДСТРАИВАЙТЕСЬ ПОД СИСТЕМУ! СВЕРНИТЕ С РЕЛЬС!! БУНТ!!! АНТИХАЙП!11!!1
    const unsigned int* buffer1,
          unsigned int* buffer2,
          unsigned int* buffer3,
    const unsigned int n,
    const unsigned int k)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n) return;

    buffer2[index] = 1 - ((buffer1[index] >> k) & 1);
    buffer3[index] = (buffer1[index] >> k) & 1;
}

namespace cuda {
void radix_sort_01_local_counting(const gpu::WorkSize &workSize,
            const gpu::gpu_mem_32u &buffer1, gpu::gpu_mem_32u &buffer2,
            gpu::gpu_mem_32u &buffer3, unsigned int n, unsigned int k)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::radix_sort_01_local_counting<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(
        buffer1.cuptr(), buffer2.cuptr(), buffer3.cuptr(), n, k);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
