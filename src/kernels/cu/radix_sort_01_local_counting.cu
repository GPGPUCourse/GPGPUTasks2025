#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "helpers/rassert.cu"
#include "../defines.h"

__global__ void radix_sort_01_map(
    // это лишь шаблон! смело меняйте аргументы и используемые буфера! можете сделать даже больше кернелов, если это вызовет затруднения - смело спрашивайте в чате
    // НЕ ПОДСТРАИВАЙТЕСЬ ПОД СИСТЕМУ! СВЕРНИТЕ С РЕЛЬС!! БУНТ!!! АНТИХАЙП!11!!1
    const unsigned int* input_gpu,
          unsigned int* map_result,
          unsigned int  n,
          unsigned int  bit_offset)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n)
    {
        return;
    }

    //map_result[idx] = ((input_gpu[idx] >> bit_offset) & 1) == 0;
    map_result[idx] = (input_gpu[idx] >> bit_offset) & 1;

    //implicit sync by kernel end
}

namespace cuda {
void radix_sort_01_map(const gpu::WorkSize& workSize,
            const gpu::gpu_mem_32u &buffer1, gpu::gpu_mem_32u &buffer2, unsigned int a1, unsigned int a2)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::radix_sort_01_map<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(buffer1.cuptr(), buffer2.cuptr(), a1, a2);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
