#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "helpers/rassert.cu"
#include "../defines.h"

__global__ void prefix_sum_simple(
    const unsigned int* a, // input n
          unsigned int* c, // output; n
    unsigned int n)
{
    int i = threadIdx.x;
    int glob_i = blockIdx.x * blockDim.x + i;
    __shared__ unsigned int locin[GROUP_SIZE];
    __shared__ unsigned int locout[GROUP_SIZE];

    if (glob_i < n) {
        locin[i] = a[glob_i];
    } else {
        locin[i] = 0;
    }
    locout[i] = 0;
    __syncthreads();

    // count prefix sum for WorkGroup
    int sz = 1;
    int mask = i + 1; //mask = [1; GROUP_SZ]
    while (sz <= GROUP_SIZE) { //maybe just <
        if (mask & sz) {
            mask -= sz;
            locout[i] += locin[mask];
            // printf("out[%d] += in[%d;%d) (=%u)\n", i, mask, mask + sz, locin[mask]);
        }
        __syncthreads();
        int l = i * sz;
        int r = (i + 1) * sz;
        if (r < GROUP_SIZE) { // +- ok for code divergence 
            locin[l] = locin[l] + locin[r]; // sum two blocks of size `sz`
        }
        __syncthreads();
        sz *= 2;
    }
    
    //load from local
    if (glob_i < n) {
        c[glob_i] = locout[i];;
    } 
}

namespace cuda {
void prefix_sum_simple(const gpu::WorkSize &workSize, 
    const gpu::gpu_mem_32u &a, unsigned int abase,
    gpu::gpu_mem_32u &c, unsigned int cbase,
    unsigned int n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::prefix_sum_simple<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(a.cuptr() + abase, c.cuptr() + cbase, n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
