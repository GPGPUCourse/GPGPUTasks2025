#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "helpers/rassert.cu"
#include "../defines.h"

__global__ void matrix_multiply_naive(
                       const float* a, // rows=h x cols=k
                       const float* b, // rows=k x cols=w
                             float* c, // rows=h x cols=w
                       unsigned int w,
                       unsigned int h,
                       unsigned int k)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int glob_col = index % w;
    const unsigned int glob_row = index / w;

    if (index >= w * h) {
        return;
    }

    float accum = 0.;

    for (int i = 0; i < k; ++i)
    {
        accum += a[glob_row * k + i] * b[i * w + glob_col];
    }

    c[glob_row * w + glob_col] = accum;
}

namespace cuda {
void matrix_multiply_naive(const gpu::WorkSize &workSize,
            const gpu::gpu_mem_32f &a, const gpu::gpu_mem_32f &b, gpu::gpu_mem_32f &c, unsigned int w, unsigned int h, unsigned int k)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::matrix_multiply_naive<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(a.cuptr(), b.cuptr(), c.cuptr(), w, h, k);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
