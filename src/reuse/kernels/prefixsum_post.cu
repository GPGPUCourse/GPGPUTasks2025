#include "hip/hip_runtime.h"

#include <libgpu/context.h>
#include <libgpu/work_size.h>
#include <libgpu/shared_device_buffer.h>

#include <libgpu/cuda/cu/common.cu>

#include "helpers/rassert.cu"
#include "../defines.h"
#include "../wrappers.h"

__global__ void prefixsum_post(
    unsigned int* b, // reduced buffer;
    unsigned int* c, // output
    unsigned int n)
{
    int i = threadIdx.x;
    int glob_i = blockIdx.x * blockDim.x + i;
    
    unsigned int offset = 0;
    if (blockIdx.x > 0) {
        offset = b[blockIdx.x - 1];
    }
    
    if (glob_i < n) {
        c[glob_i] = c[glob_i] + offset;
    }
}

namespace cuda {
void prefixsum_post(const gpu::WorkSize& workSize, gpuptr::u32 b, gpuptr::u32 c, unsigned int n)
{
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::prefixsum_post<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(b.cuptr(), c.cuptr(), n);
    CUDA_CHECK_KERNEL(stream);
}
} // namespace cuda
