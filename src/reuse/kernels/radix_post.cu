#include "hip/hip_runtime.h"
#include <libgpu/context.h>
#include <libgpu/shared_device_buffer.h>
#include <libgpu/work_size.h>

#include <libgpu/cuda/cu/common.cu>

#include "../defines.h"
#include "../wrappers.h"
#include "helpers/rassert.cu"

__global__ void radix_post(const unsigned int* a, const unsigned int* b, unsigned int* c, unsigned int offset, unsigned int n) {
    int x = threadIdx.x;
    int i = blockIdx.x * blockDim.x + x;
    unsigned int zero_count = b[n - 1];
    if (i < n) {
        int t = ((a[i] >> offset) & 1);
        int cind = (1 - t) * (b[i] - 1) + t * (zero_count + i - b[i]);
        c[cind] = a[i];
    }
}

namespace cuda {
void radix_post(const gpu::WorkSize& workSize, const gpuptr::u32 a, gpuptr::u32 b, gpuptr::u32 c, unsigned int offset, unsigned int n) {
    gpu::Context context;
    rassert(context.type() == gpu::Context::TypeCUDA, 34523543124312, context.type());
    hipStream_t stream = context.cudaStream();
    ::radix_post<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(a.cuptr(), b.cuptr(), c.cuptr(), offset, n);
    CUDA_CHECK_KERNEL(stream);
}
}  // namespace cuda
